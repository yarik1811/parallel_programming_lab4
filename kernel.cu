#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
__global__ void matmul1(float* a, float* b, int n, float* c)
{
    // Computed subelement
    float sum = 0.0f;

    // Start indexes calculation
    int ia = (blockDim.y * blockIdx.y + threadIdx.y) * n;//ewf
    int ib = blockDim.x * blockIdx.x + threadIdx.x;
    int ic = ia + ib;

    // Multiply two matrices
    for (int k = 0; k < n; k++)
        sum += a[ia + k] * b[ib + k * n];

    // Write the block sub-matrix to global memory;
    // each thread writes one element
    c[ic] = sum;
}

#define kernel matmul1
#include "gpu.h"